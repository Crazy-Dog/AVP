#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <iostream>
#include <Windows.h>

#define SIZEA 50000
#define SIZEB ((SIZEA+1)/2)

using namespace std;
//1.�������� ���
//2.���������
//3.�������� ����������� ������
__global__ void addKernel(const int *a, int *b)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < SIZEB) {
		b[i] = a[i] + a[SIZEA - 1 - i];
	}
}

int main()
{
    int *a = new int[SIZEA];
    int *b = new int[SIZEB];
	int *dev_a = 0;
	int *dev_b = 0;

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//random array values
	for (int i = 0; i < SIZEA; i++) {
		//srand(time(NULL));
		a[i] = rand();
	}

	// Allocate GPU buffers for three vectors (two input, one output)
	hipMalloc((void**)&dev_a, SIZEA * sizeof(int));
	hipMalloc((void**)&dev_b, SIZEB * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, SIZEA * sizeof(int), hipMemcpyHostToDevice);

	dim3 threads(512);
	dim3 blocks((SIZEB + 511) / 512);

	hipEventRecord(start, 0);
	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<blocks, threads>>>(dev_a, dev_b);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipMemcpy(b, dev_b, SIZEB * sizeof(int), hipMemcpyDeviceToHost);

	/*for (int i = 0; i < SIZEA; i++) {
		cout << a[i] << " ";
	}
	cout << "\n";
	for (int i = 0; i < SIZEB; i++) {
		cout << b[i] << " ";
	}*/

	hipEventElapsedTime(&time, start, stop);
	cout << endl << time << endl;

	system("pause");

	hipFree(dev_a);
	hipFree(dev_b);

    return 0;
}
