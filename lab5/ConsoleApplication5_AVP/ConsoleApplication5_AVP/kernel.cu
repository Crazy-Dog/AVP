#include "hip/hip_runtime.h"
#ifdef __INTELLISENSE__
void __syncthreads();
#endif
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <iostream>
#include <Windows.h>

#define SIZEA 2048
#define SIZEB ((SIZEA+1)/2)

using namespace std;

void cudaFunc(const int* a, int* b);
void cpuFunc(const int* a, int* b);
//2.������������ ������������
//���������� �����������
__global__ void addKernel(const int *a, int *b)
{
	__shared__ int tmp[1024];
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < SIZEB) {
		tmp[threadIdx.x] = a[i];
		tmp[threadIdx.x + 512] = a[SIZEA - 1 - i];
	}	

	__syncthreads();

	if (i < SIZEB) {
		b[i] = tmp[threadIdx.x] + tmp[threadIdx.x + 512];
	}
}

int main() {
	int *a = new int[SIZEA];
	int *resCuda = new int[SIZEB];
	int *resCPU = new int[SIZEB];

	for (int i = 0; i < SIZEA; i++) {
		a[i] = rand();
	}

	cudaFunc(a, resCuda);
	cpuFunc(a, resCPU);

	system("pause");
	return 0;
}

void cudaFunc(const int* a, int* b)
{
	int *dev_a = 0;
	int *dev_b = 0;

	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//random array values
	

	// Allocate GPU buffers for three vectors (two input, one output)
	hipMalloc((void**)&dev_a, SIZEA * sizeof(int));
	hipMalloc((void**)&dev_b, SIZEB * sizeof(int));

	// Copy input vectors from host memory to GPU buffers.
	hipMemcpy(dev_a, a, SIZEA * sizeof(int), hipMemcpyHostToDevice);

	dim3 threads(512);
	dim3 blocks((SIZEB + 511) / 512);

	hipEventRecord(start, 0);
	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <blocks, threads >> >(dev_a, dev_b);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	hipMemcpy(b, dev_b, SIZEB * sizeof(int), hipMemcpyDeviceToHost);

	/*for (int i = 0; i < SIZEA; i++) {
	cout << a[i] << " ";
	}
	cout << "\n";
	for (int i = 0; i < SIZEB; i++) {
	cout << b[i] << " ";
	}*/

	hipEventElapsedTime(&time, start, stop);
	cout << time << endl;

	hipFree(dev_a);
	hipFree(dev_b);
}

void cpuFunc(const int* a, int* b)
{
	double res = 0;
	LARGE_INTEGER tmp;
	QueryPerformanceCounter(&tmp);
	res = tmp.QuadPart;

	for (int i = 0; i < SIZEB; i++) {
		b[i] = a[i] + a[SIZEA - 1 - i];
	}
	QueryPerformanceCounter(&tmp);
	res = tmp.QuadPart - res;
	QueryPerformanceFrequency(&tmp);
	res /= tmp.QuadPart / 1000;

	/*for (int i = 0; i < SIZEA; i++) {
	cout << a[i] << " ";
	}
	cout << "\n";
	for (int i = 0; i < SIZEB; i++) {
	cout << b[i] << " ";
	}*/

	cout << res << endl;
}